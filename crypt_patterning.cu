#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <time.h>

#include "../yalla/include/dtypes.cuh"
#include "../yalla/include/inits.cuh"
#include "../yalla/include/polarity.cuh"
#include "../yalla/include/property.cuh"
#include "../yalla/include/solvers.cuh"
#include "../yalla/include/utils.cuh"
#include "../yalla/include/vtk.cuh"

// Generic simulation parameters
const auto r_max = 1;
const auto r_eq = 0.8;
const auto prolif_rate = 0.04f;
const auto n_0 = 2500;//5000;
const auto n_max = 1000000;
const auto dt = 0.1;
const auto force_modifier = 0.2f; // This controls the ratio between forces and friction

const auto real_time = 1000.0f;//1000.0f;
// const auto relax_time = 100.0f;
auto n_time_steps = int(real_time/dt);
auto skip_step = n_time_steps/100;

// Model parameters
const auto k_diff = 1.0f;
const auto k_stem = 1.0f;
// const auto t_diff = 4.0f; // differentiation time-scale
const auto k_w_d = 0.2f;
const auto k_b_d = 0.5f;
// const auto k_w_deg = 0.05;
const auto k_b_deg = 0.5;
const auto k_pol = 0.0f;//5.0f;

// const auto paneth_ratio = 0.05f;
const auto stem_cell_ratio = 0.8f;

const auto k_het = 2.0f;
// const auto Ft = 3.0f;
const auto compression_ratio = 0.3f;

const auto polarity_half_life = 1.0;
const auto polarity_update_amplitude = 2*M_PI;

// const auto r_seed = 1.2f;
// const auto r_pattern = 2.55f;
// const auto l_pattern = 9.24f;

std::string output_label = "crypt_simulation";
// std::string output_path = "/g/sharpeba/members/Miquel_Marin/crypt_patterning_output/crypt_patterning_sweep_time_scales_15-06-21/";
// std::string output_path = "/home/miquel/ownCloud/crypt_patterning_simulation/output/";
// std::string output_path = "/g/sharpe-hd/marin/crypt_patterning_spatial_scales_28-10-21/";
// std::string output_path = "/g/sharpe/members/Miquel_Marin/crypt_patterning_output/crypt_patterning_spatial_scales_12-01-21/";
// std::string output_path = "/g/sharpe/members/Miquel_Marin/crypt_patterning_output/crypt_patterning_sweep_time_scales_28-01-22/";
// std::string output_path = "/g/sharpe/members/Miquel_Marin/crypt_patterning_output/crypt_patterning_small_sweep_w_patterns_07-02-22/";
// std::string output_path = "/home/miquel/data/crypt_simulation_output/crypt_patterning_small_sweep_w_patterns_30-08-22/";
std::string output_path = "output";


MAKE_PT(crypt_cell, w, b, theta, phi, diff);

// command line parameters that need to be passed inside the solver device methods
#define N_PARAMS 3
__device__ float* d_solver_params;

__device__ float* d_cell_cycle;
__device__ int* d_epi_nbs;
__device__ bool* d_is_paneth;
__device__ float* d_compression;
__device__ bool* d_is_pattern;
__device__ int* d_n_homotypic;
__device__ int* d_n_heterotypic;


__device__ crypt_cell relaxation_force(
    crypt_cell Xi, crypt_cell r, float dist, int i, int j)
{
    crypt_cell dF{0};
    if (i == 0 or j == 0) return dF; // ghost node
    if (i == j) return dF;

    if (dist > r_max) return dF;

    d_epi_nbs[i]+=1;

    auto F = force_modifier * (fmaxf(r_eq - dist, 0) - fmaxf(dist - r_eq, 0));
    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    // dF.z = r.z * F / dist;

    return dF;
}

__device__ crypt_cell force(
    crypt_cell Xi, crypt_cell r, float dist, int i, int j)
{
    crypt_cell dF{0};
    if (i == 0 or j == 0) return dF; // ghost node
    if (i == j){
        auto effective_w = Xi.w + float(d_is_pattern[i]);
        dF.w = d_is_paneth[i] - d_solver_params[2]*Xi.w;
        dF.b = (effective_w <= 0.05) - k_b_deg*Xi.b ;
        // dF.b = (Xi.diff >= 1.0) - k_b_deg*Xi.b ;

        // dF.b = 1.0 * (Xi.diff>= 0.2f) - 0.1*Xi.b;
        auto pos_inc = k_diff * Xi.b;
        auto neg_inc = k_stem * effective_w;

        if(Xi.diff > 1.0){
            pos_inc = 0.0f;
            neg_inc = 0.0f;
        } else if(Xi.diff < 0.0)
            neg_inc = 0.0f;

        auto t_diff = d_solver_params[0];
        dF.diff = (pos_inc - neg_inc)/t_diff;

        return dF;
    }

    if (dist > r_max) return dF;

    auto is_homotypic = false;
    if(abs(r.diff)<0.5)
        is_homotypic = true;
    auto k_adh = 1.0;
    auto k_rep = 1.0;
    if(!is_homotypic)
        k_rep = k_het;

    auto F = force_modifier * (k_rep*fmaxf(r_eq - dist, 0) - k_adh*fmaxf(dist - r_eq, 0));

    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    // dF.z = r.z * F / dist;

    dF.w = -k_w_d*r.w;
    dF.b = -k_b_d*r.b;

    d_epi_nbs[i] += 1;
    if(is_homotypic)
        d_n_homotypic[i] += 1;
    else
        d_n_heterotypic[i] += 1;

    // Biasing polarity vector via heterospecific contacts (eph-ephrin like repulsion)
    auto r_hat = pt_to_pol(-r, dist);
    // auto diff_j = Xi.diff - r.diff;
    dF -= !is_homotypic * k_pol*unidirectional_polarization_force(Xi, r_hat);

    if(r_eq > dist)
        d_compression[i] += r_eq - dist;


    return dF;
}


__global__ void proliferate(float rate, int n_cells, hiprandState* d_state,
    crypt_cell* d_X, float3* d_old_v, int* d_n_cells)
{
    D_ASSERT(n_cells * rate <= n_max);
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells) return;  // Dividing new cells is problematic!
    if (i == 0) return;  // ghost node

    if(d_X[i].diff >= 1.0) return;

    if(d_compression[i]/float(d_epi_nbs[i]) > compression_ratio*r_eq) return;

    if(!d_is_paneth[i])
        d_cell_cycle[i] += rate * dt;


    if (d_cell_cycle[i] < 1.f) return;

    auto n = atomicAdd(d_n_cells, 1);
    // auto theta = acosf(2. * hiprand_uniform(&d_state[i]) - 1);
    auto phi = hiprand_uniform(&d_state[i]) * 2 * M_PI;
    // d_X[n].x = d_X[i].x + r_eq / 4 * sinf(theta) * cosf(phi);
    // d_X[n].y = d_X[i].y + r_eq / 4 * sinf(theta) * sinf(phi);
    // d_X[n].z = d_X[i].z + r_eq / 4 * cosf(theta);
    d_X[n].x = d_X[i].x + 0.25*r_eq * cosf(phi);
    d_X[n].y = d_X[i].y + 0.25*r_eq * sinf(phi);
    d_X[n].z = 0.0;
    d_old_v[n] = d_old_v[i];

    d_X[i].w = 0.5*d_X[i].w;
    d_X[i].b = 0.5*d_X[i].b;
    d_X[n].w = d_X[i].w;
    d_X[n].b = d_X[i].b;
    d_X[n].diff = d_X[i].diff;

    d_X[n].theta = d_X[i].theta;
    d_X[n].phi = hiprand_uniform(&d_state[i]) * 2 * M_PI;

    d_is_paneth[n] = d_is_paneth[i];
    d_compression[n] = d_compression[i];
    d_is_paneth[n] = false;

    d_cell_cycle[i] = -0.25 + 0.5*hiprand_uniform(&d_state[i]);
    d_cell_cycle[n] = -0.25 + 0.5*hiprand_uniform(&d_state[n]);

}


__global__ void set_up_cell_cycle(int* d_n_cells, crypt_cell* d_X, hiprandState* d_state)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    d_cell_cycle[i] = hiprand_uniform(&d_state[i]);
}


__global__ void update_pattern(const int n_cells, crypt_cell* d_X, float r_pattern, float l_pattern)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>= n_cells) return;

    d_is_pattern[i] = false;

    auto px = d_X[i].x;
    auto py = d_X[i].y;

    auto center0_x = floor(px / l_pattern) * l_pattern;
    auto center1_x = ceil(px / l_pattern) * l_pattern;

    auto center0_y= floor(py / l_pattern) * l_pattern;
    auto center1_y = ceil(py / l_pattern) * l_pattern;
    // printf("i %i x %f x0 %f x1 %f\n",i, d_X[i].x, center0_x, center1_x);

    auto in_pattern = false;
    // distance from center 0-0
    auto dist = sqrt(pow(px - center0_x, 2) + pow(py - center0_y, 2));
    if(dist < r_pattern)
        in_pattern = true;

    // distance from center 0-1
    dist = sqrt(pow(px - center0_x, 2) + pow(py - center1_y, 2));
    if(dist < r_pattern)
        in_pattern = true;

    // distance from center 1-0
    dist = sqrt(pow(px - center1_x, 2) + pow(py - center0_y, 2));
    if(dist < r_pattern)
        in_pattern = true;

    // distance from center 1-1
    dist = sqrt(pow(px - center1_x, 2) + pow(py - center1_y, 2));
    if(dist < r_pattern)
        in_pattern = true;

    if(in_pattern){
        d_is_pattern[i] = true;
        // d_X[i].w -= w_diff*(d_X[i].w - 1.0);
    }
}

// Implementation of cell motility ********************************************

__global__ void update_polarities(const int n_cells, crypt_cell* d_X,
    float prob_update, hiprandState* d_state)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>= n_cells) return;

    if (hiprand_uniform(&d_state[i]) < prob_update){
        d_X[i].phi += polarity_update_amplitude * (hiprand_uniform(&d_state[i]) - 0.5);
        if (d_X[i].phi < 0.0)
            d_X[i].phi = 2 * M_PI + d_X[i].phi;
        else if(d_X[i].phi > 2*M_PI)
            d_X[i].phi = d_X[i].phi - 2 * M_PI;

    }

}

template<typename Pt>
using Traction_force = void(const Pt* __restrict__ d_X, const int i,
    Pt* d_dX);

template<typename Pt>
__device__ void constant_force_on_vector(const Pt* __restrict__ d_X, const int i,
    Pt* d_dX)
{
        if(i == 0) return;
        if(d_X[i].diff >= 1.0) return;
        auto Ft = d_solver_params[1];
        auto phi = d_X[i].phi;
        d_dX[i].x += cosf(phi) * force_modifier * Ft;
        d_dX[i].y += sinf(phi) * force_modifier * Ft;
}

template<typename Pt, Traction_force<Pt> force>
__global__ void traction(const Pt* __restrict__ d_X, Pt* d_dX,
    int n_max)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_max) return;

    force(d_X, i, d_dX);
}

template<typename Pt, Traction_force<Pt> force>
void traction_forces(const int n, const Pt* __restrict__ d_X, Pt* d_dX)
{
    traction<Pt, force><<<(n + 32 - 1) / 32, 32>>>(
        d_X, d_dX, n);
}

//*****************************************************************************

int main(int argc, const char* argv[])
{
    // Prepare initial state
    Solution<crypt_cell, David_Gabriel_solver> cells{n_max, 50, 1.0f};

    Property<float> solver_params{N_PARAMS, "solver_params"};
    hipMemcpyToSymbol(HIP_SYMBOL(
        d_solver_params), &solver_params.d_prop, sizeof(d_solver_params));


    auto there_is_pattern = std::stoi(argv[1]) == 1;  // command line argument: 0 for no-pattern, 1 for pattern

    auto Ft = std::stof(argv[2]);  // stem cell traction force
    auto t_diff = std::stof(argv[3]);  // cell differentiation time scale
    auto r_seed = std::stof(argv[4]);  // mean cell-cell distance at t0
    auto k_w_deg = std::stof(argv[5]);  // degradation rate of Wnt
    auto paneth_ratio = std::stof(argv[6]);  // Initial ratio of Paneth cells
    auto r_pattern = std::stof(argv[7]);  // radius of Wnt patterns
    auto l_pattern = std::stof(argv[8]);  // radius of Wnt patterns


    std::string replicate = argv[9];

    solver_params.h_prop[0] = t_diff;
    solver_params.h_prop[1] = Ft;
    solver_params.h_prop[2] = k_w_deg;
    solver_params.copy_to_device();

    // Polarity vector used for cell motility randomly changes orientation
    // with exponential probability defined by a half-life parameter
    auto pol_update_probability = log(2)*dt/polarity_half_life;

    if(!there_is_pattern)
        output_label += "_no_pattern";
    else
        output_label += "_w_pattern";

    output_label = output_label + "_Ft_" + argv[2] +
                "_t-diff_" + argv[3] +
                "_r-seed_" + argv[4] +
                "_k-w-deg_" + argv[5] +
                "_paneth-ratio_" + argv[6] +
                "_r-pattern_" + argv[7] +
                "_l-pattern_" + argv[8] +
                "_rep_" + argv[9];

    // output_label += "_Ft_"+std::to_string(Ft).substr(0,5);

    std::cout<<output_label<<std::endl;


    *cells.h_n = n_0;
    random_disk(r_seed, cells);


    Property<float> cell_cycle{n_max,"cell_cycle"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_cell_cycle), &cell_cycle.d_prop, sizeof(d_cell_cycle));

    Property<bool> is_paneth{n_max,"is_paneth"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_is_paneth), &is_paneth.d_prop, sizeof(d_is_paneth));

    Property<bool> is_pattern{n_max,"is_pattern"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_is_pattern), &is_pattern.d_prop, sizeof(d_is_pattern));


    Property<float> compression{n_max,"compression"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_compression), &compression.d_prop, sizeof(d_compression));


    cells.h_X[0].x = 0.0f;
    cells.h_X[0].y = 0.0f;
    cells.h_X[0].z = 0.0f;
    is_paneth.h_prop[0] = false;
    cells.h_X[0].w = 0.0f;
    cells.h_X[0].b = 0.0f;
    cells.h_X[0].diff = 0.0f;
    cell_cycle.h_prop[0] = 0.0f;
    cells.set_fixed(0);

    for (auto i = 1; i < n_0; i++) {
        auto temp = cells.h_X[i].z;
        cells.h_X[i].z = cells.h_X[i].x;
        cells.h_X[i].x = temp;
        cell_cycle.h_prop[i] = rand() / (RAND_MAX + 1.);

        cells.h_X[i].theta = 0.5*M_PI;
        cells.h_X[i].phi = rand() / (RAND_MAX + 1.) * 2 * M_PI;

        cells.h_X[i].diff = 0.0;
        cells.h_X[i].w = 0.0f;
        cells.h_X[i].b = 0.0f;

        is_paneth.h_prop[i] = false;
        auto dice = rand() / (RAND_MAX + 1.);
        if (dice < paneth_ratio)
            is_paneth.h_prop[i] = true;
        else if (dice < stem_cell_ratio)
            cells.h_X[i].diff = 0.0;
        else
            cells.h_X[i].diff = 1.0;

    }
    cells.copy_to_device();
    cell_cycle.copy_to_device();
    is_paneth.copy_to_device();

    Property<int> n_epi_nbs{n_max,"n_epi_nbs"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));

    Property<int> n_homotypic{n_max,"n_homotypic"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_n_homotypic), &n_homotypic.d_prop, sizeof(d_n_homotypic));

    Property<int> n_heterotypic{n_max,"n_heterotypic"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_n_heterotypic), &n_heterotypic.d_prop, sizeof(d_n_heterotypic));

    auto traction = [&](const int n, const crypt_cell* __restrict__ d_X, crypt_cell* d_dX) {
        thrust::fill(thrust::device, n_epi_nbs.d_prop,
            n_epi_nbs.d_prop + cells.get_d_n(), 0);
        thrust::fill(thrust::device, compression.d_prop,
            compression.d_prop + cells.get_d_n(), 0);

        thrust::fill(thrust::device, n_homotypic.d_prop,
            n_homotypic.d_prop + cells.get_d_n(), 0);
        thrust::fill(thrust::device, n_heterotypic.d_prop,
            n_heterotypic.d_prop + cells.get_d_n(), 0);

        return traction_forces<crypt_cell, constant_force_on_vector>(n, d_X, d_dX);
    };

    // // Relaxation phase
    // for (auto time_step = 0; time_step <= int(relax_time/dt); time_step++)
    //     cells.take_step<relaxation_force, friction_on_background>(dt, reset_nbs);


    hiprandState* d_state;
    hipMalloc(&d_state, n_max * sizeof(hiprandState));
    auto seed = time(NULL);
    setup_rand_states<<<(n_max + 128 - 1) / 128, 128>>>(n_max, seed, d_state);

    set_up_cell_cycle<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(cells.d_n, cells.d_X, d_state);

    // Simulate growth
    Vtk_output output{output_label, output_path, true};
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {

        if(there_is_pattern)
            update_pattern<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
                cells.get_d_n(), cells.d_X, r_pattern, l_pattern);

        update_polarities<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
            cells.get_d_n(), cells.d_X, pol_update_probability, d_state);

        cells.take_step<force, friction_on_background>(dt, traction);
        proliferate<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
            prolif_rate, cells.get_d_n(), d_state,
            cells.d_X, cells.d_old_v, cells.d_n);

        if(time_step % skip_step == 0){
            hipDeviceSynchronize();
            cells.copy_to_host();
            is_paneth.copy_to_host();
            is_pattern.copy_to_host();
            compression.copy_to_host();
            n_homotypic.copy_to_host();
            n_heterotypic.copy_to_host();


            n_epi_nbs.copy_to_host();
            output.write_positions(cells);
            output.write_polarity(cells);
            output.write_field(cells, "w", &crypt_cell::w);
            output.write_field(cells, "b", &crypt_cell::b);
            output.write_field(cells, "diff", &crypt_cell::diff);
            output.write_property(n_epi_nbs);
            output.write_property(is_paneth);
            output.write_property(is_pattern);
            output.write_property(compression);
            output.write_property(n_homotypic);
            output.write_property(n_heterotypic);


        }

    }

    return 0;
}
